/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <time.h>
#include <fstream>
#include <vector>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>

float L,LL; int N, C,itera;

using namespace std;

// función Maxwelliana de la distribución de las partículas.
__device__ float distribution (float vb, float aleatorio, hiprandState *states)     //generador de distribución maxwelliana para la velocidad
{

  // Genera un valor random v
   float fmax = 0.5 * (1.0 + exp (-2.0 * vb * vb));
   float vmin = - 5.0 * vb;
   float vmax = + 5.0 * vb;
   float v;
   float f;
   float x;
   int Idx = blockIdx.x*blockDim.x + threadIdx.x;

   while(true){
	   v = vmin + ((vmax - vmin) * aleatorio);
	   f = 0.5 * (exp (-(v - vb) * (v - vb) / 2.0) +
			    exp (-(v + vb) * (v + vb) / 2.0));
	   x = fmax * aleatorio;
	   if(x > f) aleatorio = hiprand_uniform(states + Idx);
	   else return v;
   }

}
//Distribución aleatoria de las partículas.
__global__ void distribucionParticulas(float *rx,float *ry,float *vx,float *vy,int N,hiprandState *states,float vb,float L){
	int Idx = blockIdx.x*blockDim.x + threadIdx.x;

	unsigned int seed = (unsigned int) (clock() * Idx);
	hiprand_init(seed, 0, 0, states + Idx);

	if(Idx < N){
		 rx[Idx] = L*hiprand_uniform(states + Idx);    //inicializando la posicion aleatoria en x
		 ry[Idx] = L*hiprand_uniform(states + Idx);
		 vx[Idx] = distribution(vb,hiprand_uniform(states + Idx),states);//;L*curand_uniform_float(states + Idx);//distribution(vb,states);                          //inicializa la velocidad con una distribucion maxwelliana
		 vy[Idx] = distribution(vb,hiprand_uniform(states + Idx),states);//L*curand_uniform_float(states + Idx);//distribution(vb,states);                          //inicializa la velocidad con una distribucion maxwelliana

	}


}
// inicialización de la densidad.
__global__ void inicializacionDensidad(float *ne,int C){
	int Id=blockIdx.x*blockDim.x + threadIdx.x;
		if(Id<(C*C)){
			ne[Id]=0.0;
		}
 }


//Calculo de la densidad en cada celda.

__global__ void calculoDensidad(float *rx, float *ry, float *ne, int N, int C,float L){
	int Id=blockIdx.x*blockDim.x + threadIdx.x;
	 float dx = L / float (C);
	 float dxx=L/float(C*C);
	if(Id<N){

				int jx = int(rx[Id]/dx); //posicion en x de la particula
			    int jy = int(ry[Id]/dx); //posicion en y de la particula
			    float yx = (rx[Id]/dx) - (float)jx; //posicion exacta de la particula en x de la celda "j"
			    //float yy = (ry[Id]/dx) - (float)jy; //posicion exacta de la particula en y de la celda "j"
			    ne[(jy*C)+jx] += (1. - yx)/dxx;
			    if(jx+1==C) ne[(jy*C)] += yx/dxx;
			    else ne[(jy*C)+jx+1] += yx/dxx;

    }

}
//pasar de reales a complejos.

__global__ void real2complex (float *ne, hipfftComplex *u, int C)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
	int index =idy*C+idx;

	if ( idx < C && idy <C)
	 {
		u[index].x = ne[index];
		u[index].y = 0.0f;
	}

}
//__global__ void prueba (cufftComplex *vf, float *vr, int C){
//	int idx = blockIdx.x*blockDim.x+threadIdx.x;
//	int idy = blockIdx.y*blockDim.y+threadIdx.y;
//	int index =idy*C+idx;
//
//	if(idx<C && idy<C){
//
//		vr[index]= (vf[index].x)/((float)C*(float)C*(float)C*(float)C);
//		vr[index]= (vf[index].y)/((float)C*(float)C*(float)C*(float)C);
//
//	}
//}

__global__ void solve_Poisson(hipfftComplex *vf, hipfftComplex *v, int C,float L){

	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
	float dx = L / float (C);
	float i,W,Wm,Wn;
	i = (0.0,L);
	W = exp(2.0 * M_PI * i / float(C));
	Wm = L;
	Wn = L;
	if(idx<C && idy<C){
		int index = idy*C+idx;
		float denom;
		denom = 4.0;
		denom -= (Wm + (L / Wm) + Wn +( L / Wn));
		if (denom != 0.0){
				vf[index].x *= dx*dx/denom;
				vf[index].y *= dx*dx/denom;
				}
				Wn *= W;//se multiplica por la constante W
			}
			Wm *= W;
			if(idx<C && idy<C){
			int index = idx*C+idy;
			v[index].x=vf[index].x;
			v[index].y=vf[index].y;
			}

	}



__global__ void complex2real(hipfftComplex *v, float *vr, int C){
	/* compute idx and idy, the location of the element in the original CxC array */
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int idy = blockIdx.y*blockDim.y+threadIdx.y;
	if ( idx < C && idy <C)
		 {
		 int index = idy*C+idx;
		 vr[index] = v[index].x /((float)C*(float)C);
		 }
}



////////////////////////////////////////////////////////////////////////////////////////////////////
int main(){
	// Parametros
	L = 64.0;            // dominio de la solucion 0 <= x <= L (en longitudes de debye)
	//L=LL*LL;
	N = 10000;            // Numero de particulas
	C = 64;            // Número de celdas.
	float vb = 3.0;    // velocidad promedio de los electrones
	//float kappa = 2. * M_PI / (L);
	//float dt=0.1;    // delta tiempo (en frecuencias inversas del plasma)
	//float tmax=10000;  // cantidad de iteraciones. deben ser 100 mil segun el material
	//int skip = int (tmax / dt) / 10; //saltos del algoritmo para reportar datos
	//int itera=0;
	float salida=0.;
	 //float dx = L / float (C);

/////////////////////////////////////////////////////////////////////////////////////////////////////
//Inicializacion de la posición de las particulas en x, y y velocidad en vx,vy del host y dispositivo.
	float *rx_h,*ry_h,*vx_h,*vy_h;
	float *rx_d,*ry_d, *vx_d,*vy_d;
////////////////////////////////////////////////////////////////////////////////////////////////////
	// inicialización de las variables de densidad del host y dispositivo.
	float *ne_h;
	float *ne_d;
	float *vr_h;
	float *vr_d;
////////////////////////////////////////////////////////////////////////////////////////////////////
	//inicializacion tipo complex a real.

	hipfftComplex *u_complex_d,*vf_complex_d,*v_complex_d ;
	hipMalloc((void**)&u_complex_d,sizeof(hipfftComplex)*C*C);
	hipMalloc((void**)&vf_complex_d,sizeof(hipfftComplex)*C*C);
	hipMalloc((void**)&v_complex_d,sizeof(hipfftComplex)*C*C);

////////////////////////////////////////////////////////////////////////////////////////////////////
	int size = N*sizeof(float);
	int size_ne=C*C*sizeof(float);

//////////////////////////////////////////////////////////////////////////////////////////////////////
	//reserva en memoria al host
	rx_h = (float *)malloc(size);
	ry_h = (float *)malloc(size);
	vx_h = (float *)malloc(size);
	vy_h = (float *)malloc(size);
	ne_h = (float *)malloc(size_ne);
	vr_h = (float *)malloc(size_ne);

//////////////////////////////////////////////////////////////////////////////////////////////////////
	//reserva de memoria del dispositivo.
	hipMalloc((void **)&rx_d,size);
	hipMalloc((void **)&ry_d,size);
	hipMalloc((void **)&vx_d,size);
	hipMalloc((void **)&vy_d,size);
	hipMalloc((void **)&ne_d,size_ne);
	hipMalloc((void **)&vr_d,size_ne);

////////////////////////////////////////////////////////////////////////////////////////////////////

	//valores aleatorios y tamaños de los vectores.
	hiprandState *devStates;
	hipMalloc((void **) &devStates, N * sizeof(hiprandState));


	float blockSize = 1024;
	dim3 dimBlock (ceil(N/blockSize), 1, 1);
	dim3 dimBlock2 (ceil((C*C)/blockSize), 1, 1);
	dim3 dimGrid (blockSize, 1, 1);


	distribucionParticulas<<<blockSize,dimBlock>>>(rx_d,ry_d,vx_d,vy_d,N,devStates,vb,L);
	hipDeviceSynchronize();

	inicializacionDensidad<<<blockSize,dimBlock2>>>(ne_d,C);
	hipDeviceSynchronize();

	calculoDensidad<<<blockSize,dimBlock>>>(rx_d,ry_d,ne_d,N,C,L);
	hipDeviceSynchronize();

	hipfftHandle plan;
	hipfftPlan2d(&plan, C, C, HIPFFT_C2C);

	real2complex<<<blockSize,dimBlock2>>>(ne_d,u_complex_d,C);
	hipDeviceSynchronize();

	hipfftExecC2C (plan, u_complex_d, vf_complex_d, HIPFFT_FORWARD);
	// dividir el resultado por C4
	//prueba<<<dimGrid, dimBlock2>>> (vf_complex_d,vr_d,C);

	v_complex_d[0].x=0.0;
	v_complex_d[0].y=0.0;


	solve_Poisson<<<dimGrid, dimBlock2>>> (vf_complex_d,v_complex_d,C,L);
	hipDeviceSynchronize();

	hipfftExecC2C (plan, v_complex_d, v_complex_d, HIPFFT_BACKWARD);

	complex2real<<<dimGrid, dimBlock2>>> (v_complex_d,vr_d,C);


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	//posición en x.
	hipMemcpy(rx_h, rx_d, size, hipMemcpyDeviceToHost);

	// posición en y.
	hipMemcpy(ry_h, ry_d, size, hipMemcpyDeviceToHost);

	// velocidad en x.
	hipMemcpy(vx_h, vx_d, size, hipMemcpyDeviceToHost);

	//velocidad en y.
	hipMemcpy(vy_h, vy_d, size, hipMemcpyDeviceToHost);
	//inicializacion densidades
	hipMemcpy(ne_h, ne_d, size_ne, hipMemcpyDeviceToHost);
	//calculo poisson
	hipMemcpy (vr_h , vr_d, size_ne, hipMemcpyDeviceToHost);

	///////////////////Imprimir los resultados en archivos//////////////////////
	ofstream init;
		init.open("distribucionInicial.txt");
		  		    for (int i = 0; i < N; i++){
		  		    	init<<rx_h[i]<<" "<<ry_h[i]<<" "<<vx_h[i]<<" "<<vy_h[i]<<endl;

		  		    }

		  		    init.close();


		init.open("salida_densidad3.txt");
					for (int i = 0; i < C*C; i++){
						init<<ne_h[i]<<endl;
						salida+=ne_h[i];
					}

					init.close();
					cout<<salida<<" "<<endl;

		init.open("entrada_poisson");
					for (int i = 0; i < C; i++){
						for (int j = 0; j < C; j++){
								init<<ne_h[(C*i)+j]<<" ";
						}
						init<<endl;
					}
					init.close();

		init.open("poisson");
				for (int i = 0; i < C; i++){
					for (int j = 0; j < C; j++){
						init<< vr_h[(C*j)+i]<<" ";
							}
							init<<endl;
						}
						init.close();

////////////////////Liberar memoria//////////////////////////
	free(rx_h);
	free(ry_h);
	free(vx_h);
	free(vy_h);
	free(ne_h);
	free(vr_h);
	hipfftDestroy(plan);
	hipFree(rx_d);
	hipFree(ry_d);
	hipFree(vx_d);
	hipFree(vy_d);
	hipFree(ne_d);
	hipFree(vr_d);
	hipFree(u_complex_d);
	hipFree(vf_complex_d);
	hipFree(v_complex_d);
	return (0);

}
